#include "hip/hip_runtime.h"
#include <stdio.h>

__global__ void kernel() {
    printf("Hello from kernel\n");
}

int main() {
    kernel << <1, 1 >> > ();
    hipDeviceSynchronize();
    return 0;
}